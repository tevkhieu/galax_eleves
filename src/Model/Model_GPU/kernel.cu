
#include <hip/hip_runtime.h>
#ifdef GALAX_MODEL_GPU

#include "cuda.h"
#include "kernel.cuh"
#define DIFF_T (0.1f)
#define EPS (1.0f)

__global__ void compute_acc(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU, float* massesGPU, int n_particles)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	float3 position_i = positionsGPU[i];
	float3 acceleration_i = {0.0f, 0.0f, 0.0f};
	float3 diff_pos;
	float rdij;

	for (int j = 0; j < n_particles; ++j) {
		diff_pos.x = positionsGPU[j].x - position_i.x;
		diff_pos.y = positionsGPU[j].y - position_i.y;
		diff_pos.z = positionsGPU[j].z - position_i.z;

		rdij = (diff_pos.x * diff_pos.x + diff_pos.y * diff_pos.y + diff_pos.z * diff_pos.z);
		rdij = rsqrtf((rdij * rdij * rdij));
        rdij = fminf(10.0 * rdij, 10.0);
		rdij = rdij * massesGPU[j];

		acceleration_i.x += diff_pos.x * rdij;
		acceleration_i.y += diff_pos.y * rdij;
		acceleration_i.z += diff_pos.z * rdij;
	}

	accelerationsGPU[i] = acceleration_i;

}

__global__ void maj_pos(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	float3 velocities = velocitiesGPU[i];
	float3 particles = positionsGPU[i];
	float3 accelerations = accelerationsGPU[i];
	velocities.x += accelerations.x * 2.0f;
	velocities.y += accelerations.y * 2.0f;
	velocities.z += accelerations.z * 2.0f;
	particles.x += velocities.x * 0.1f;
	particles.y += velocities.y * 0.1f;
	particles.z += velocities.z * 0.1f;

	positionsGPU[i] = particles;
	velocitiesGPU[i] = velocities;

}

void update_position_cu(float3* positionsGPU, float3* velocitiesGPU, float3* accelerationsGPU, float* massesGPU, int n_particles)
{
	int nthreads = 32;
	int nblocks =  (n_particles + (nthreads -1)) / nthreads;

	compute_acc<<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU, massesGPU, n_particles);
	maj_pos    <<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU);
}


#endif // GALAX_MODEL_GPU